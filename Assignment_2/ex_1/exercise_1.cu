
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void cuda_hello(){
    printf("Hello World! My thread ID is %d\n\n", threadIdx.x);
}

int main() {
    
    cuda_hello<<<1,256>>>(); 
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    return 0;
}

