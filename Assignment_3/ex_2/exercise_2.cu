
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <iostream>
#include <chrono>


struct Particle {
  float3 position;
  float3 velocity;
};

__host__ __device__ float3 velocity_update(float3 velocity, float time) {
  float3 u_vel;

  u_vel.x = velocity.x + sin(time);
  u_vel.y = velocity.y + sin(time);
  u_vel.z = velocity.z + sin(time);

  return u_vel;
}

__global__ void gpu_step(int n, Particle *particles, float time) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n) {
    particles[i].velocity = velocity_update(particles[i].velocity, time);

    particles[i].position.x += particles[i].velocity.x;
    particles[i].position.y += particles[i].velocity.y;
    particles[i].position.z += particles[i].velocity.z;
  }
}

void cpu_step(int n, Particle *particles, float time) {
  for (int i = 0; i < n; i++) {
    particles[i].velocity = velocity_update(particles[i].velocity, time);

    particles[i].position.x += particles[i].velocity.x;
    particles[i].position.y += particles[i].velocity.y;
    particles[i].position.z += particles[i].velocity.z;
  }
}

float rand_float() {
  return (float)(rand()) / ((float)(RAND_MAX));
}

void init_particles(int n, Particle *particles) {
  for (int i = 0; i < n; i++) {
    float3 pos, vel;

    pos.x = rand_float();
    pos.y = rand_float();
    pos.z = rand_float();

    vel.x = rand_float();
    vel.y = rand_float();
    vel.z = rand_float();

    particles[i].position = pos;
    particles[i].velocity = vel;
  }
}

double mse_difference(int n, Particle *xp, Particle *yp) {
  double mse = 0;
  for (int i = 0; i < n; i++) {
    mse += (xp[i].position.x - yp[i].position.x) * (xp[i].position.x - yp[i].position.x);
    mse += (xp[i].position.y - yp[i].position.y) * (xp[i].position.y - yp[i].position.y);
    mse += (xp[i].position.z - yp[i].position.z) * (xp[i].position.z - yp[i].position.z);

    mse += (xp[i].velocity.x - yp[i].velocity.x) * (xp[i].velocity.x - yp[i].velocity.x);
    mse += (xp[i].velocity.y - yp[i].velocity.y) * (xp[i].velocity.y - yp[i].velocity.y);
    mse += (xp[i].velocity.z - yp[i].velocity.z) * (xp[i].velocity.z - yp[i].velocity.z);

  }

  return mse / (double)n;
}

void print_particles(int n, Particle *par) {
  for (int i = 0; i < n; i++) {
    std::cout << "n: " << i << " px: " << par[i].position.x << " py: " << par[i].position.y
      << " pz: " << par[i].position.z << " vx: " << par[i].velocity.x << " vy: " << par[i].velocity.y
      << " vz: " << par[i].velocity.z << std::endl;
  }
}

int main(int argc, char **argv) {
  int n_par = std::atoi(argv[1]), n_it =std::atoi(argv[2]), block_size = std::atoi(argv[3]);

  hipError_t err;

  Particle *par_host, *par_device, *par_device_result_on_host;


  par_host = (Particle*)malloc(sizeof(Particle) * n_par);
  init_particles(n_par, par_host);

  // Initialize memory that will contain the GPU particles on host
  // Copy the CPU particles to the GPU particles
  err = hipHostMalloc(&par_device_result_on_host, sizeof(Particle) * n_par, hipHostMallocDefault);
  if (err != hipSuccess) {
    printf("Error %s", hipGetErrorString(err));
  }

  memcpy(par_device_result_on_host, par_host, sizeof(Particle) * n_par);


  float t = 0;

  // Simulate on CPU
  auto start = std::chrono::steady_clock::now();
  //for (int i = 0; i < n_it; i++) {
    //cpu_step(n_par, par_host, t);

    //t += 1.0;
  //}

  auto end = std::chrono::steady_clock::now();

  //std::cout << "CPU " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() << std::endl;


  // reset time
  t = 0;

  // Allocate device memory 
  hipMalloc(&par_device, sizeof(Particle)*n_par);

  // Simulate on GPU
  start = std::chrono::steady_clock::now();
  for (int i = 0; i < n_it; i++) {
    // At the beginning of timestep copy from host to device
    hipMemcpy(par_device, par_device_result_on_host, sizeof(Particle) * n_par, hipMemcpyHostToDevice);

    // Perform one update
    gpu_step<<<(n_par / block_size) + 1, block_size>>>(n_par, par_device, t);

    // Copy from devince to host
    hipMemcpy(par_device_result_on_host, par_device, sizeof(Particle) * n_par, hipMemcpyDeviceToHost);
    //err = cudaDeviceSynchronize();
    //if (err != cudaSuccess) {
    //printf("Error %s", cudaGetErrorString(err));
    //}

    t += 1.0;
  }

  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("Error %s", hipGetErrorString(err));
  }

  end = std::chrono::steady_clock::now();

  std::cout << "GPU " << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() << std::endl;


  //std::cout << "Particles from GPU:\n";
  //print_particles(n_par, par_device_result_on_host);
  //std::cout << "\n";

  //std::cout << "Particles from CPU:\n";
  //print_particles(n_par, par_host);
  //std::cout << "\n";

  //double mse = mse_difference(n_par, par_device_result_on_host, par_host);
  //std::cout << "GPU - CPU mean squared error: " << mse << std::endl;


  // Free memory
  free(par_host);
  hipFree(par_device);

  return 0;
}
